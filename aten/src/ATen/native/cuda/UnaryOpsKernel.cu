#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>

namespace at { namespace native {

void bitwise_not_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}

void logical_not_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, iter.dtype(1), "logical_not_cuda", [&]() {
    using self_t = scalar_t;
    AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, iter.dtype(0), "logical_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(self_t a) -> scalar_t { return static_cast<scalar_t>(!a); });
    });
  });
}

void ceil_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "ceil_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::ceil(a);
    });
  });
}

void floor_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "floor_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::floor(a);
    });
  });
}

void neg_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND(ScalarType::Half, iter.dtype(), "neg_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return -a;
    });
  });
}

void round_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "round_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      // We do not use std::round because we would like to round midway numbers to the nearest even integer.
      return std::nearbyint(a);
    });
  });
}

void trunc_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "trunc_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::trunc(a);
    });
  });
}

// overloading rsqrt and rsqrtf
template <typename scalar_t>
__host__ __device__ static inline scalar_t rsqrt_wrapper(scalar_t a) {
  return static_cast<scalar_t>(::rsqrtf(static_cast<float>(a)));
}

__host__ __device__ static inline double rsqrt_wrapper(double a) {
  return ::rsqrt(a);
}

void rsqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "rsqrt_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return rsqrt_wrapper(a);
    });
  });
}

void sign_kernel_cuda(TensorIterator& iter){
    if (iter.dtype() == ScalarType::Bool) {
      gpu_kernel(iter, []GPU_LAMBDA(bool a){
        return a;
      });
    } else {
      AT_DISPATCH_ALL_TYPES_AND(ScalarType::Half, iter.dtype(), "sign_cuda", [&]() {
          gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
              scalar_t zero = scalar_t(0);
              return (zero < a) - (a < zero);
          });
      });
    }
}

void erfinv_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return erfinvf(a);
    });
  });
}

REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);
REGISTER_DISPATCH(logical_not_stub, &logical_not_kernel_cuda);
REGISTER_DISPATCH(ceil_stub, &ceil_kernel_cuda);
REGISTER_DISPATCH(floor_stub, &floor_kernel_cuda);
REGISTER_DISPATCH(neg_stub, &neg_kernel_cuda);
REGISTER_DISPATCH(round_stub, &round_kernel_cuda);
REGISTER_DISPATCH(rsqrt_stub, &rsqrt_kernel_cuda);
REGISTER_DISPATCH(sign_stub, &sign_kernel_cuda);
REGISTER_DISPATCH(trunc_stub, &trunc_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
}}
